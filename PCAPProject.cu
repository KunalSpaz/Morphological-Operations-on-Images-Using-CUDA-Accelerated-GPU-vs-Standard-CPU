#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void dilationKernel(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int maxVal = 0;
    for (int j = -1; j <= 1; ++j) {
        for (int i = -1; i <= 1; ++i) {
            int nx = x + i;
            int ny = y + j;
            if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                unsigned char val = input[ny * width + nx];
                if (val > maxVal) maxVal = val;
            }
        }
    }
    output[y * width + x] = maxVal;
}

__global__ void erosionKernel(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int minVal = 255;
    for (int j = -1; j <= 1; ++j) {
        for (int i = -1; i <= 1; ++i) {
            int nx = x + i;
            int ny = y + j;
            if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                unsigned char val = input[ny * width + nx];
                if (val < minVal) minVal = val;
            }
        }
    }
    output[y * width + x] = minVal;
}

unsigned char* loadPGM(const char* filename, int* width, int* height) {
    FILE* fp = fopen(filename, "rb");
    if (!fp) {
        perror("Error opening file");
        return NULL;
    }

    char format[3];
    fscanf(fp, "%2s", format);
    if (format[0] != 'P' || format[1] != '5') {
        printf("Unsupported format: %s\n", format);
        fclose(fp);
        return NULL;
    }

    int maxval;
    fscanf(fp, "%d %d %d", width, height, &maxval);
    fgetc(fp); // Consume the newline after maxval

    int imgSize = (*width) * (*height);
    unsigned char* data = (unsigned char*)malloc(imgSize);
    fread(data, sizeof(unsigned char), imgSize, fp);
    fclose(fp);
    return data;
}

void savePGM(const char* filename, unsigned char* data, int width, int height) {
    FILE* fp = fopen(filename, "wb");
    fprintf(fp, "P5\n%d %d\n255\n", width, height);
    fwrite(data, sizeof(unsigned char), width * height, fp);
    fclose(fp);
}

void checkCudaError(const char* msg) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error (%s): %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    const char* filename = "image.pgm";
    int width, height;
    unsigned char* h_input = loadPGM(filename, &width, &height);
    if (!h_input) return 1;

    savePGM("input_saved.pgm", h_input, width, height); // Save input to verify

    int imgSize = width * height;
    unsigned char *d_input, *d_dilated, *d_eroded;
    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_dilated, imgSize);
    hipMalloc(&d_eroded, imgSize);

    hipMemcpy(d_input, h_input, imgSize, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + 15) / 16, (height + 15) / 16);

    dilationKernel<<<gridSize, blockSize>>>(d_input, d_dilated, width, height);
    checkCudaError("Dilation Kernel");

    erosionKernel<<<gridSize, blockSize>>>(d_input, d_eroded, width, height);
    checkCudaError("Erosion Kernel");

    hipDeviceSynchronize();

    unsigned char* h_dilated = (unsigned char*)malloc(imgSize);
    unsigned char* h_eroded = (unsigned char*)malloc(imgSize);
    hipMemcpy(h_dilated, d_dilated, imgSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_eroded, d_eroded, imgSize, hipMemcpyDeviceToHost);

    printf("Sample INPUT pixel values:\n");
    for (int i = 0; i < 10; ++i)
        printf("%d ", h_input[i]);
    printf("\n");

    printf("Sample DILATED pixel values:\n");
    for (int i = 0; i < 10; ++i)
        printf("%d ", h_dilated[i]);
    printf("\n");

    printf("Sample ERODED pixel values:\n");
    for (int i = 0; i < 10; ++i)
        printf("%d ", h_eroded[i]);
    printf("\n");

    savePGM("dilated.pgm", h_dilated, width, height);
    savePGM("eroded.pgm", h_eroded, width, height);

    hipFree(d_input);
    hipFree(d_dilated);
    hipFree(d_eroded);
    free(h_input);
    free(h_dilated);
    free(h_eroded);

    printf("Done.\n");
    return 0;
}
